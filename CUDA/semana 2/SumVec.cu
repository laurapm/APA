#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


//PROGRAMA PARA REALIZAR LA SUMA DE UN VECTOR EN LA GPU
//USANDO 3 BLOQUES CON 8 HILOS POR BLOQUE

__global__ void sumKernel(int* c, int* a, int *b, const int tamVec){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < tamVec)
		c[i] = a[i] + b[i];
}

//funcion auxiliar que usa CUDA para sumar vectores en paralelo
void auxSum(int* c, int* a, int* b, const int tamVec){
	int*  dev_a = nullptr;
	int* dev_b = nullptr;
	int* dev_c = nullptr;
	//Reservamos memoria
	
	hipMalloc((void**)&dev_a, tamVec * sizeof(int));
	hipMalloc((void**)&dev_b, tamVec * sizeof(int));
	hipMalloc((void**)&dev_c, tamVec * sizeof(int));

	//Copiamos los vectores en la GPU
	hipMemcpy(dev_a, a, tamVec * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, tamVec * sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 3;
	int threadsPerBlock = 8;
	sumKernel << <numBlocks, threadsPerBlock >> > (dev_c, dev_a, dev_a, tamVec);

	//Copiamos el resultado de la GPU en el host
	hipMemcpy(c, dev_c, tamVec * sizeof(int), hipMemcpyDeviceToHost);

	//liberamos la memoria
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

int main(int argc, char** argv) {
	const int tamVec = 24;
	int a[tamVec] = { 1, 2, 3, 4, 5, 6,7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24 };
	int b[tamVec] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24 };
	int c[tamVec] = { 0 };

	auxSum(c, a, b, tamVec);

	printf("[");
	for (int i = 0; i < tamVec; i++){
		printf("%d ", a [i]);
	}

	printf("] + [");

	for (int i = 0; i < tamVec; i++){
		printf("%d ", b[i]);
	}

	printf("] = ["); 

	for (int i = 0; i < tamVec; i++){
		printf("%d ", c[i]);
	}
	printf("] \n");

	printf("\n pulse INTRO para finalizar");
	//limpiamos el buffer
	fflush(stdin);
	char intro = getchar();
	return 0;
}

