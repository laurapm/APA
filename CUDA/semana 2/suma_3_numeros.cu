
#include <hip/hip_runtime.h>
/*

#include <iostream>
#include <stdlib.h>
#include <cuda_runtime.h>

//funcion llamadada por el host y ejecutada por el device
__global__ void suma(int a, int b, int c, int *resultado){
	*resultado = a + b + c;
}

//funcion llamada y ejecutada por el host --> __host__

int main(int argc, char ** argv){C:\Users\Laura\AppData\Local\Temp\Rar$DIa0.869\book.h
	int n1 = 3, n2 = 5, n3 = 3;
	int *hst_c, *dev_c;

	//reserva de memoria para el host y el device
	hst_c = (int*)malloc(sizeof(int));
	cudaMalloc((void**)&dev_c, sizeof(int));

	//llamada a la funcion del kernel, pasandole los datos
	suma << <1, 1 >> >(n1, n2, n3, dev_c);

	//copiamos los datos del device al host
	cudaMemcpy(hst_c, dev_c, sizeof(int), cudaMemcpyDeviceToHost);

	printf("El resultado de la operacion es \n%2d + %2d + %2d = %2d \n", n1, n2, n3, *hst_c);

	printf("\n pulse INTRO para finalizar");
	//limpiamos el buffer
	fflush(stdin);
	char intro = getchar();
	return 0;
}

*/