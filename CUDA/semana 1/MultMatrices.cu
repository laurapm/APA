#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <iostream>

#define TILE_WIDTH 8


__global__ void MulMatrices(int *dev_a, int *dev_b, int *dev_c, int Width){
	int fila = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int columna = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int suma = 0;

	for (int i = 0; i < Width; i++)
		suma += dev_a [fila * Width + i] * dev_b[i * Width + columna];

	dev_c[fila * Width + columna] = suma;
}

int main(){
	//definimos los punteros del host y de la GPU (device)
	int *host_a, *host_b, *host_c, *dev_a, *dev_b, *dev_c;

	int numBlocks = 8;
	int threadsPerBlock = 64;
	
	int contador = 0; //lo usaremos para rellenar las matrices

	//Reservamos la memoria del host y de la GPU

	int tam = numBlocks * threadsPerBlock * sizeof(int);

	host_a = (int *)malloc(tam);
	host_b = (int *)malloc(tam);
	host_c = (int *)malloc(tam);

	hipMalloc((void**)&dev_a, tam);
	hipMalloc((void**)&dev_b, tam);
	hipMalloc((void**)&dev_c, tam);

	//Rellenamos las matrices, cada fila con numeros del 0 al 15
	for (int n = 0; n< threadsPerBlock*numBlocks; n++)
	{
		host_a[n] = host_b[n] = contador;
		contador++;
		contador = contador % 16;
	}

	dim3 dimGrid(numBlocks / 2, numBlocks / 2);
	dim3 dimBlock(threadsPerBlock / TILE_WIDTH, threadsPerBlock / TILE_WIDTH);

	//Copiamos la informacion del host a la GPU
	hipMemcpy(dev_a, host_a, tam, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, tam, hipMemcpyHostToDevice);

	MulMatrices << < dimGrid, dimBlock >> > (dev_a, dev_b, dev_c, TILE_WIDTH * 2);

	//Devolvemos la informacion al host
	hipMemcpy(host_c, dev_c, tam, hipMemcpyDeviceToHost);

	//Imprimimos la matriz de resultado

	for (int i = 0; i<numBlocks * 2; i++)
	{
		for (int j = 0; j < numBlocks * 2; j++)
			printf("%d ", host_c[numBlocks * 2 * i + j]);
		printf("\n");
	}

	//liberamos memoria
	hipFree(dev_a);
	hipFree(host_a);

	printf("\n pulse INTRO para finalizar");

	//limpiamos el buffer
	fflush(stdin);
	char intro = getchar();
	return 0;
}