#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
int main(void) {
	hipDeviceProp_t prop;
	int count;
	//Comprobamos el numero de dispositivos que hay en el equipo
	HANDLE_ERROR(hipGetDeviceCount(&count));
	//Recogemos los datos de todos los dispositivos en el equipo (1 en este caso)
	for (int i = 0; i< count; i++) {
		//Imprimimos los datos de la tarjeta
		printf("Numero de dispositivos en el equipo %d; dispositivo mostrado: %d\n", count, i + 1);
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf(" --- Informacion general del dispositivo %d ---\n", i+1);
		printf("Nombre: %s\n", prop.name);
		printf("Capacidad computacion: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Timeout de ejecucion del kernel: ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf(" --- Informacion de memoria del dispositivo %d ---\n", i+1);
		printf("Memoria global total: %ld\n", prop.totalGlobalMem);
		printf("Memoria constante total: %ld\n", prop.totalConstMem);
		printf("Memoria pitch maxima: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf(" --- Informacion MP del dispositivo %d ---\n", i+1);
		printf("Numero multiprocesadores: %d\n",
			prop.multiProcessorCount);
		printf("Memoria compartida por MP: %ld\n", prop.sharedMemPerBlock);
		printf("Registros por MP: %d\n", prop.regsPerBlock);
		printf("Hilos por warp: %d\n", prop.warpSize);
		printf("Hilos maximos por bloque: %d\n",
			prop.maxThreadsPerBlock);
		printf("Dimension maxima de hilos: (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Dimension grid maxima: (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2]);
		printf("\n");
		system("PAUSE");
	}
}