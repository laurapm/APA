#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <conio.h>
#include <Windows.h>
#include <iostream>
#include <time.h>
#include <fstream>

using namespace std;
//Funciones que van a usarse a lo largo de la ejecuci�n del programa
//CPU
void generarTablero(int *tablero, int filas, int columnas, int dificultad);
void imprimirTablero(int *tablero, int filas, int columnas);
void imprimirColumnas(int columnas);
void modoManual(int *tablero, int filas, int columnas, int dificultad);
void guardarPartida(int *tablero, int nFilas, int nCols, int nDificultad);
void cargarPartida();
//GPU
__global__ void ToyBlastManual(int *tablero, int filas, int columnas, int fila, int columna, int bomba);
__device__ void compruebaPiezas(int * tablero, int columna, int fila, int filas, int columnas, int anterior);

__device__ void compruebaArriba(int *tablero, int columna, int fila, int filas, int columnas, int anterior);
__device__ void compruebaAbajo(int *tablero, int columna, int fila, int filas, int columnas, int anterior);

__device__ void compruebaDerecha(int *tablero, int columna, int fila, int filas, int columnas, int anterior);
__device__ void compruebaIzquierda(int *tablero, int columna, int fila, int filas, int columnas, int anterior);

__device__ void compruebaArribaDerecha(int *tablero, int columna, int fila, int filas, int columnas, int anterior);
__device__ void compruebaAbajoDerecha(int *tablero, int columna, int fila, int filas, int columnas, int anterior);

__device__ void compruebaArribaIzquierda(int *tablero, int columna, int fila, int filas, int columnas, int anterior);
__device__ void compruebaAbajoIzquierda(int *tablero, int columna, int fila, int filas, int columnas, int anterior);


int main(void) {
	//Almacenamos las propiedades de la tarjeta para no exceder el numero de hilos posibles en el tablero
	int *tablero;
	int filas = 0;
	int columnas = 0;
	int dificultad = 0;
	//Preguntamos si quiere cargar un juego guardado anteriormente o si quiere empezar de nuevo
	cout << "Quieres continuar una partida anterior o empezar de nuevo? (c: cargar / n: nueva partida)\n";
	char partida = 'a';
	cin >> partida;
	while (partida != 'c' && partida != 'n') {
		cout << "Introduce un valor valido para iniciar el juego\n";
		cin >> partida;
	}
	if (partida == 'n') {
		//Recogemos los datos de filas y columnas del tablero que vamos a usar
		cout << "Seleccione el numero de filas con las que desea jugar: \n";
		cin >> filas;
		cout << "Seleccione el numero de columnas con las que desea jugar: \n";
		cin >> columnas;
		//Tablero m�nimo de 8 por 8
		while (filas < 8) {
			cout << "El numero de filas con las que desea jugar es demasiado peque�o, el minimo aceptado es 8: \n";
			cin >> filas;
		}
		while (columnas < 8) {
			cout << "El numero de columnas con las que desea jugar es demasiado peque�o, el minimo aceptado es 8: \n";
			cin >> columnas;
		}
		//Seleccionamos el nivel de dificultad
		cout << "Seleccione el nivel de dificultad: \n";
		cin >> dificultad;
		//Si no entra dentro de los dos niveles que se recogen en la pr�ctica los pedimos en bucle hasta que se cumpla 
		while (dificultad != 1 && dificultad != 2) {
			cout << "Seleccione el nivel de dificultad, solo puede elegirse 1 o 2: \n";
			cin >> dificultad;
		}
		//Reservamos la memoria del tablero y lo inicializamos con generar tablero
		tablero = new int[filas * columnas];
		generarTablero(tablero, filas, columnas, dificultad);
		modoManual(tablero, filas, columnas, dificultad);
	}
	else {
		cargarPartida();
	}
	system("PAUSE");
}


//Generamos el tablero con n�meros aleatorios en funci�n de la dificultad
void generarTablero(int *tablero, int filas, int columnas, int dificultad) {
	srand(time(0));
	for (int i = 0; i < (filas * columnas); i++) {
		if (dificultad == 1) {
			tablero[i] = rand() % 5 + 1;
		}
		else {
			tablero[i] = rand() % 6 + 1;
		}
	}
}


//Rellenar tablero cuando hemos explotado bloques
void rellenarTablero(int *tablero, int filas, int columnas, int dificultad) {
	srand(time(0));
	for (int i = 0; i < (filas * columnas); i++) {
		if (tablero[i] == 0) {
			switch (dificultad) {
			case 1:
				tablero[i] = rand() % 5 + 1;
				break;
			case 2:
				tablero[i] = rand() % 6 + 1;
				break;
			}
		}
	}
}

//Funci�n que imprime el n�mero de columnas que va a tener el tablero para que sea m�s facil elegir piezas
void imprimirColumnas(int columnas) {
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         " << i + 1;
		}
		else {
			if (i < 9) {
				cout << "    " << i + 1;
			}
			else {
				cout << "   " << i + 1;
			}
		}
	}
	cout << "\n";
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         |";
		}
		else {
			cout << "    |";
		}
	}
	cout << "\n";
}
//Imprimimos el tablero
void imprimirTablero(int *tablero, int filas, int columnas) {
	cout << "SE HAN GENERADO " << filas << " FILAS Y " << columnas << " COLUMNAS\n";
	cout << "+-+-+-TABLERO DE JUEGO-+-+-+\n\n";
	imprimirColumnas(columnas);
	for (int i = 0; i < filas; i++) {
		if (i < 9) {
			cout << i + 1 << "    - ";
		}
		else {
			cout << i + 1 << "   - ";
		}
		for (int k = 0; k < columnas; k++) {
			//Damos color en funci�n del n�mero imprimido
			int bloque = tablero[i * filas + k];
			switch (bloque) {
			case 1:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
				break;
			case 2:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 4);
				break;
			case 3:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 11);
				break;
			case 4:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 2);
				break;
			case 5:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 5);
				break;
			case 6:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 6);
				break;
			case 7:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 3);
				break;
			case 8:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 3);
				break;
			case 9:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 3);
				break;
			}
			cout << "| " << bloque << " |";
		}
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
		cout << "\n";
	}
}

__device__ void compruebaPiezas(int * tablero, int columna, int fila, int filas, int columnas, int anterior)
{
	//compruebaPiezas(tablero, columnaHilo, filaHilo, filas, columnas, anterior);
	//Aqu� vamos a indicarle hacia donde tiene que buscar en funci�n de la posici�n del tablero en la cual nos encontremos
	//Primero comprobamos que no sea un tipo de bomba, en nuestro caso, las bombas van a ser 7 8 y 9 
	//7 elimina la fila, 8 la columna y 9 es el TNT
	if (tablero[(fila * columnas) + columna] != 7 && tablero[(fila * columnas) + columna] != 8 && tablero[(fila * columnas) + columna] != 9) {
		//EMPEZAMOS CON LAS PIEZAS DE LAS CUATRO ESQUINAS
		//SI ESTAMOS EN LA SUPERIOR IZQUIERDA SOLO PODEMOS COMPROBAR HACIA ABAJO, DERECHA Y DIAGONAL DERECHA
		if (fila == 0 && columna == 0) {
			compruebaDerecha(tablero, columna, fila, filas, columnas, anterior);
			compruebaAbajo(tablero, columna, fila, filas, columnas, anterior);
		}
		//SI ESTAMOS EN LA SUPERIOR DERECHA SOLO PODEMOS COMPROBAR HACIA ABAJO, IZQUIERDA Y DIAGONAL IZQUIERDA
		if (fila == 0 && columna == (columnas - 1)) {
			compruebaIzquierda(tablero, columna, fila, filas, columnas, anterior);
			compruebaAbajo(tablero, columna, fila, filas, columnas, anterior);
		}
		//SI ESTAMOS EN LA INFERIOR IZQUIERDA SOLO PODEMOS COMPROBAR HACIA ARRIBA, DERECHA Y DIAGONAL DERECHA
		if (fila == (filas - 1) && columna == 0) {
			compruebaDerecha(tablero, columna, fila, filas, columnas, anterior);
			compruebaArriba(tablero, columna, fila, filas, columnas, anterior);
		}
		//SI ESTAMOS EN LA INFERIOR DERECHA SOLO PODEMOS COMPROBAR HACIA ARRIBA, IZQUIERDA Y DIAGONAL IZQUIERDA
		if (fila == (filas - 1) && columna == (columnas - 1)) {
			compruebaArriba(tablero, columna, fila, filas, columnas, anterior);
			compruebaIzquierda(tablero, columna, fila, filas, columnas, anterior);
		}
		//UNA VEZ COMPROBADAS LAS ESQUINAS, AUN TENEMOS OTROS CUATRO CASOS ESPECIALES, ESTAR EN LA FILA DE ARRIBA, FILA DE ABAJO, COLUMNA DE LA IZQ Y COLUMNA DE LA DERECHA
		//SI ESTAMOS EN LA FILA DE ARRIBA SOLO PODEMOS IR HACIA IZQ, DERECHA, DIAGONAL DERECHA, DIAGONAL IZQUIERDA Y HACIA ABAJO
		if (fila == 0) {
			compruebaIzquierda(tablero, columna, fila, filas, columnas, anterior);
			compruebaDerecha(tablero, columna, fila, filas, columnas, anterior);
			compruebaAbajo(tablero, columna, fila, filas, columnas, anterior);
		}
		//SI ESTAMOS EN LA FILA DE ABAJO SOLO PODEMOS IR HACIA IZQ, DERECHA, DIAGONAL DERECHA, DIAGONAL IZQUIERDA Y ARRIBA
		if (fila == (filas - 1)) {
			compruebaIzquierda(tablero, columna, fila, filas, columnas, anterior);
			compruebaDerecha(tablero, columna, fila, filas, columnas, anterior);
			compruebaArriba(tablero, columna, fila, filas, columnas, anterior);
		}
		//SI ESTAMOS EN LA COLUMNA IZQUIERDA SOLO SE COMPRUEBA HACIA DERECHA, ARRIBA, ABAJO, DIAGONAL DERECHA Y DIAGONAL IZQ
		if (columna == 0) {
			compruebaDerecha(tablero, columna, fila, filas, columnas, anterior);
			compruebaArriba(tablero, columna, fila, filas, columnas, anterior);
			compruebaAbajo(tablero, columna, fila, filas, columnas, anterior);

		}
		//SI ESTAMOS EN LA COLUMNA DERECHA SOLO SE COMPRUEBA HACIA IZQUIERDA, ARRIBA, ABAJO, DIAGONAL DERECHA Y DIAGONAL IZQ
		if (columna == (columnas - 1)) {
			compruebaIzquierda(tablero, columna, fila, filas, columnas, anterior);
			compruebaArriba(tablero, columna, fila, filas, columnas, anterior);
			compruebaAbajo(tablero, columna, fila, filas, columnas, anterior);

		}
		//CUALQUIER OTRO CASO
		else {
			compruebaArriba(tablero, columna, fila, filas, columnas, anterior);
			compruebaAbajo(tablero, columna, fila, filas, columnas, anterior);
			compruebaDerecha(tablero, columna, fila, filas, columnas, anterior);
			compruebaIzquierda(tablero, columna, fila, filas, columnas, anterior);
		}
	}
	else { //BOMBAS
		//7 elimina la fila, 8 la columna y 9 es el TNT
		if (tablero[(fila * columnas) + columna] == 7) {
			compruebaDerecha(tablero, columna, fila, filas, columnas, 7);
			compruebaIzquierda(tablero, columna, fila, filas, columnas, 7);
		}
		else if (tablero[(fila * columnas) + columna] == 8) {
			compruebaAbajo(tablero, columna, fila, filas, columnas, 8);
			compruebaArriba(tablero, columna, fila, filas, columnas, 8);
		}
		else if (tablero[(fila * columnas) + columna] == 9) {
			compruebaAbajo(tablero, columna, fila, filas, columnas, 9);
			compruebaArriba(tablero, columna, fila, filas, columnas, 9);
			compruebaDerecha(tablero, columna, fila, filas, columnas, 9);
			compruebaIzquierda(tablero, columna, fila, filas, columnas, 9);
			compruebaAbajoDerecha(tablero, columna, fila, filas, columnas, 9);
			compruebaAbajoIzquierda(tablero, columna, fila, filas, columnas, 9);
			compruebaArribaDerecha(tablero, columna, fila, filas, columnas, 9);
			compruebaArribaIzquierda(tablero, columna, fila, filas, columnas, 9);
		}
	}
}


__device__ void compruebaArriba(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	if (anterior == 8) {
		for (int i = 0; (fila - i) >= 0; i++) {
			tablero[((fila - i) * columnas) + columna] = 0;
		}
	}
	else if (anterior == 9) {
		tablero[(fila * columnas) + columna] = 0;
		if (fila != 0) {
			tablero[((fila - 1) * columnas) + columna] = 0;
		}
	}
	else {
		if (tablero[((fila - 1) * columnas) + columna] == anterior) {
			tablero[((fila - 1) * columnas) + columna] = 0;
			tablero[(fila * columnas) + columna] = 0;
			compruebaPiezas(tablero, columna, fila - 1, filas, columnas, anterior);
		}
	}
}

__device__ void compruebaAbajo(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	if (anterior == 8) {
		for (int i = 0; (fila + i) < filas; i++) {
			tablero[((fila + i) * columnas) + columna] = 0;
		}
	}
	else if (anterior == 9) {
		tablero[(fila * columnas) + columna] = 0;
		if (fila != (filas - 1)) {
			tablero[((fila + 1) * columnas) + columna] = 0;
		}
	}
	else {
		if (tablero[((fila + 1) * columnas) + columna] == anterior) {
			tablero[((fila + 1) * columnas) + columna] = 0;
			tablero[(fila * columnas) + columna] = 0;
			compruebaPiezas(tablero, columna, fila + 1, filas, columnas, anterior);
		}
	}
}

__device__ void compruebaDerecha(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	if (anterior == 7) {
		for (int i = 0; (fila + i) < columnas; i++) {
			tablero[(fila  * columnas) + i] = 0;
		}
	}
	else if (anterior == 9) {
		tablero[(fila * columnas) + columna] = 0;
		if (columna != (columnas - 1)) {
			tablero[(fila * columnas) + columna + 1] = 0;
		}
	}
	else {
		if (tablero[(fila * columnas) + (columna + 1)] == anterior) {
			tablero[(fila * columnas) + (columna + 1)] = 0;
			tablero[(fila * columnas) + columna] = 0;
			compruebaPiezas(tablero, columna + 1, fila, filas, columnas, anterior);
		}
	}
}

__device__ void compruebaIzquierda(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	if (anterior == 7) {
		for (int i = 0; (fila - i) >= 0; i++) {
			tablero[(fila  * columnas) - i] = 0;
		}
	}
	else if (anterior == 9) {
		tablero[(fila * columnas) + columna] = 0;
		if (columna != 0) {
			tablero[(fila * columnas) + columna - 1] = 0;
		}
	}
	else {
		if (tablero[(fila * columnas) + (columna - 1)] == anterior) {
			tablero[(fila * columnas) + (columna - 1)] = 0;
			tablero[(fila * columnas) + columna] = 0;
			compruebaPiezas(tablero, columna - 1, fila, filas, columnas, anterior);
		}
	}
}

__device__ void compruebaArribaDerecha(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	//en columna = columnas -1 y en fila = 0
	if (columna != columnas - 1 && fila != 0) {
		tablero[((fila - 1) * columnas) + columna + 1] = 0;
	}
}
__device__ void compruebaAbajoDerecha(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	//en columna = columnas - 1 y en fila = filas - 1
	if (columna != columnas - 1 && fila != filas - 1) {
		tablero[((fila + 1) * columnas) + columna + 1] = 0;
	}
}

__device__ void compruebaArribaIzquierda(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	//en columna = 0 y en fila = 0
	if (columna != 0 && fila != 0) {
		tablero[((fila - 1) * columnas) + columna - 1] = 0;
	}
}
__device__ void compruebaAbajoIzquierda(int *tablero, int columna, int fila, int filas, int columnas, int anterior) {
	//en columna = 0 y en fila = filas -1
	if (columna != 0 && fila != filas - 1) {
		tablero[((fila + 1) * columnas) + columna - 1] = 0;
	}
}


__global__ void ToyBlastManual(int *tablero, int filas, int columnas, int columna, int fila, int bomba) {
	__shared__ int Tablero_Compartida[5000];
	//Recogemos la fila y la columna del hilo
	int columnaHilo = blockIdx.y * blockDim.y + threadIdx.y;
	int filaHilo = blockIdx.x*blockDim.x + threadIdx.x;
	//int posy = blockIdx.x * blockDim.x + threadIdx.x;
	//int posx = blockIdx.y * blockDim.y + threadIdx.y;
	int coordenadaHilo = columnaHilo*columnas + filaHilo;
	int ficha = fila * columnas + columna;

	Tablero_Compartida[coordenadaHilo] = tablero[coordenadaHilo];
	__syncthreads();
	int contador = 0;
	__syncthreads();
	if (coordenadaHilo == ficha) {
		int anterior = Tablero_Compartida[coordenadaHilo];
		compruebaPiezas(Tablero_Compartida, columnaHilo, filaHilo, filas, columnas, anterior);
		for (int i = 0; i < filas * columnas; i++) {
			if (Tablero_Compartida[i] == 0) {
				contador++;
			}
		}
		if (contador >= 6 && anterior != 9 && anterior != 7 && anterior != 8) {
			Tablero_Compartida[ficha] = 9;
		}
		if (contador == 5) {
			Tablero_Compartida[ficha] = bomba; //Tengo que pasarle la bomba ya generada porque con hiprand me descuadraba todas las comprobaciones
		}
	}
	__syncthreads();
	//Sube los ceros que hemos colocado al comprobar la posicion pedida por teclado bajando hacia abajo los bloques
	if (columnaHilo < columnas && filaHilo < filas) {
		if (columnaHilo < columnas&& filaHilo < filas) {
			for (int i = 1; i < filas; i++) {

				if (tablero[(filas - i)*columnas + columnaHilo] == 0) {
					if (tablero[(filas - (i + 1))*columnas + columnaHilo] == 0) {
						int j = i;
						while (tablero[(filas - (j + 1))*columnas + columnaHilo] == 0 && j < filas) {
							j++;
						}
						tablero[(filas - i)*columnas + columnaHilo] = tablero[(filas - (j + 1))*columnas + columnaHilo];
						tablero[(filas - (j + 1))*columnas + columnaHilo] = 0;
					}
					else {
						tablero[(filas - i)*columnas + columnaHilo] = tablero[(filas - (i + 1))*columnas + columnaHilo];
						tablero[(filas - (i + 1))*columnas + columnaHilo] = 0;
					}
				}
				__syncthreads();

			}
		}
	}
	__syncthreads();
	tablero[coordenadaHilo] = Tablero_Compartida[coordenadaHilo];

}

void guardarPartida(int *tablero, int filas, int columnas, int dificultad) {
	ofstream doc;
	doc.open("partida.txt");
	doc << filas << "\n";
	doc << columnas << "\n";
	doc << dificultad << "\n";
	for (int i = 0; i < filas * columnas; i++) {
		doc << tablero[i] << " ";
	}
	doc.close();
	system("cls");
	cout << "Guardado correctamente.\n\n";
}

void cargarPartida() {
	const string fichero = "partida.txt";
	ifstream leer;
	leer.open(fichero.c_str());
	int  d, *tablero;
	int i = 0;
	int n = 48;
	int f = 0;
	int c = 0;
	char fila[80];
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		while (n > 47 && n < 58) {
			n = (int)fila[i];
			i++;
			if (n > 47 && n < 58) {
				f = f * 10 + (n - 48);
			}
		}

	}
	n = 48;
	i = 0;
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		while (n > 47 && n < 58) {
			n = (int)fila[i];
			i++;
			if (n > 47 && n < 58) {
				c = c * 10 + (n - 48);
			}
		}

	}
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		d = (int)fila[0] - 48;
	}


	tablero = new int[f*c];
	for (int i = 0; i < f * c; i++) {
		leer.getline(fila, 80, ' ');
		tablero[i] = (int)fila[0] - 48;
	}
	leer.close();
	modoManual(tablero, f, c, d);
}

//Modo de juego manual
void modoManual(int *tablero, int filas, int columnas, int dificultad) {
	system("cls");
	hipError_t cudaStatus;
	int fila = 1, columna = 1;
	while (fila != 0 || columna != 0) {
		system("cls");
		imprimirTablero(tablero, filas, columnas);
		cout << "Introduce la fila en la que esta la ficha que deseas eliminar (0 para salir): \n";
		cin >> fila;
		while (fila < 0 && fila > filas) {
			cout << "Numero de fila no valido, introduzca uno en rango 1 - " << filas << ":\n";
			cin >> fila;
		}
		cout << "Introduce la columna en la que esta la ficha que deseas eliminar (0 para salir): \n";
		cin >> columna;
		while (columna < 0 && columna > columnas) {
			cout << "Numero de columna no valido, introduzca uno en rango 1 - " << columnas << ":\n";
			cin >> columna;
		}
		//Aqui empieza la fiesta con CUDA. 
		int *tablero_gpu;
		//Reservamos memoria y copiamos el tablero en la GPU
		hipMalloc((void**)&tablero_gpu, (filas * columnas) * sizeof(int));
		hipMemcpy(tablero_gpu, tablero, (filas * columnas) * sizeof(int), hipMemcpyHostToDevice);
		int mayor = 0;
		if (filas > columnas) {
			mayor = filas;
		}
		else {
			mayor = columnas;
		}
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		int TILE_WIDTH = (int)sqrt((float)deviceProp.maxThreadsPerBlock);
		int nbloques = (mayor + TILE_WIDTH - 1) / TILE_WIDTH;
		dim3 DimGrid(nbloques, nbloques);
		dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);
		ToyBlastManual << < DimGrid, DimBlock >> > (tablero_gpu, filas, columnas, columna - 1, fila - 1, (rand() % 3) + 7);
		cudaStatus = hipGetLastError();
		if (cudaStatus == hipSuccess) {
			hipMemcpy(tablero, tablero_gpu, sizeof(int)* filas * columnas, hipMemcpyDeviceToHost);
			rellenarTablero(tablero, filas, columnas, dificultad);
			hipFree(tablero_gpu);
		}
		else {
			fprintf(stderr, "Fallo en kernel\n");
			break;
		}
	}
	if (cudaStatus == hipSuccess) {
		system("cls");
		cout << "Deseas guardar la partida? (s/n)\n";
		char guardar = 'a';
		cin >> guardar;
		while (guardar != 's' && guardar != 'n') {
			system("cls");
			cout << "Valor no valido, quieres guardar la partida? (s/n): \n";
			cin >> guardar;
		}
		if (guardar == 's') {
			guardarPartida(tablero, filas, columnas, dificultad);
		}
		else {
			cout << "Saliendo sin guardar...\n \n";
		}
	}
	else {
		system("pause");
	}
}