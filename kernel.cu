#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <conio.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <windows.h>

using namespace std;

//Funciones que van a utilizarse a lo largo del programa
//CPU
void generarTablero(int *tablero, int filas, int columnas, int dificultad);
void imprimirTablero(int *tablero, int filas, int columnas);
void imprimirColumnas(int columnas);
void comprobarLleno(int *tablero, int filas, int columnas, int dificultad, bool &salida);
void generarSemillas(int *tablero, int filas, int columnas, int dificultad);
void guardarPartida(int *tablero, int filas, int columnas, int dificultad);
void cargarPartida();
void modoManual(int *tablero, int filas, int columnas, int dificultad);

//GPU
__global__ void juegoManual(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaSemillas(int *tablero, int filas, int columnas, char movimiento);
__device__ void compruebaArriba(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaAbajo(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaDerecha(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaIzquierda(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void moverCeros(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);

int main(void){

	//Almacenamos las propiedades de la tarjeta para no exceder el numero de hilos posibles en el tablero
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	//Propiedades del tablero
	int *tablero;
	int filas = 0;
	int columnas = 0;
	int dificultad = 0;
	char modo_juego;

	//Preguntamos si quiere cargar un juego guardado anteriormente o si quiere empezar de nuevo
	cout << "Quieres continuar una partida anterior o empezar de nuevo? (C: Cargar / N: Nueva partida)\n";
	char partida = 'X';
	cin >> partida;
	while (partida != 'C' && partida != 'N') {
		cout << "Introduce un valor valido para iniciar el juego\n";
		cin >> partida;
	}
	if (partida == 'N'){
		//Recogemos los datos de filas y columnas del tablero que vamos a usar
		cout << "Seleccione el numero de filas con las que desea jugar: \n";
		cin >> filas;
		cout << "Seleccione el numero de columnas con las que desea jugar: \n";
		cin >> columnas;

		//Tablero m�nimo de 4 por 4
		while (filas < 4) {
			cout << "El numero de filas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
			cin >> filas;
		}
		while (columnas < 4) {
			cout << "El numero de columnas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
			cin >> columnas;
		}
		while (prop.maxThreadsPerBlock < (filas * columnas)) {
			cout << "Has excedido el limite de semillas posibles para el tablero, introduce las filas y las columnas de nuevo: \n";
			cout << "Seleccione el numero de filas con las que desea jugar: \n";
			cin >> filas;
			cout << "Seleccione el numero de columnas con las que desea jugar: \n";
			cin >> columnas;
		}

		cout << "Elija dificultad: \n1. Bajo, se lanzaran 15 semillas de 2, 4 y 8 \n"
			"2. Dificil, se lanzaran 8 semillas de 2 y 4 \n";
		cin >> dificultad;


		while (!(dificultad == 1 || dificultad == 2)){
			cout << "Dificultad no v�lida \n";
			cout << "Selecccione 1 si desea jugar con nivel o 2 si desea jugar con nivel dificil \n";
			cin >> dificultad;
		}

		cout << "Elija modo de juego: \n A. Autom�tico \n M. Manual \n";
		cin >> modo_juego;
		while (!(modo_juego == 'M' || modo_juego == 'A')){
			cout << "Modo de juego no v�lido \n";
			cout << "Selecccione A para jugar en modo autom�tico o M para manual \n";
			cin >> modo_juego;
		}

		//Reservamos la memoria del tablero y lo inicializamos con generar tablero
		tablero = new int[filas * columnas];
		generarTablero(tablero, filas, columnas, dificultad);
		modoManual(tablero, filas, columnas, dificultad);

	}
	else {
		cargarPartida();
	}
	system("PAUSE");
}

//Generar tablero con n�meros aleatorios
void generarTablero(int *tablero, int filas, int columnas, int dificultad){
	srand(time(0));
	int tama�o = filas * columnas;
	for (int i = 0; i < tama�o; i++){
		tablero[i] = 0;
	}
	generarSemillas(tablero, filas, columnas, dificultad);
}

void comprobarLleno(int *tablero, int filas, int columnas, int dificultad, bool &salida){

	int tama�o = filas * columnas;
	int contador, posicion = 0;
	if (dificultad == 1){
		contador = 15;
		while (contador > 0 && posicion < tama�o){
			if (tablero[posicion] == 0) contador--;
			posicion++;
		}
		if (contador == 0) generarSemillas(tablero, filas, columnas, dificultad);
		else{
			cout << "Juego terminado\n";
			//exit(0);
			salida = true;
		}
	}
	if (dificultad == 2){
		contador = 8;
		while (contador > 0 && posicion < tama�o){
			if (tablero[posicion] == 0) contador--;
			posicion++;
		}
		if (contador == 0) generarSemillas(tablero, filas, columnas, dificultad);
		else{
			cout << "Juego terminado\n";
			//exit(0);
			salida = true;
		}
	}

}

//Genera los n�meros para jugar en el tablero
void generarSemillas(int *tablero, int filas, int columnas, int dificultad){
	if (dificultad == 1){
	int semillas = 0;
	int valores[3] = { 2, 4, 8 };
	while (semillas < 15){
	int posicion = rand() % (filas*columnas + 1);
	int valor = rand() % 3;
	if (tablero[posicion] == 0){
	tablero[posicion] = valores[valor];
	semillas++;
	}
	}
	}
	if (dificultad == 2){
	int semillas = 0;
	int valores[3] = { 2, 4 };
	while (semillas < 8){
	int posicion = rand() % (filas*columnas + 1);
	int valor = rand() % 2;
	if (tablero[posicion] == 0){
	tablero[posicion] = valores[valor];
	semillas++;
	}
	}
	}/*
	int tama�o = filas * columnas;
	int contador = 0;
	while (contador < 3){
		int aux = rand() % 3;
		int i = rand() % tama�o;
		if (tablero[i] == 0){
			switch (aux){
			case 0:
				tablero[i] = 2;
				break;
			case 1:
				tablero[i] = 4;
				break;
			case 2:
				tablero[i] = 8;
				break;
			}
			contador++;
		}
	}*/
}

//Funci�n que imprime el n�mero de columnas que va a tener el tablero para que sea m�s facil elegir semillas
void imprimirColumnas(int columnas) {
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         " << i + 1;
		}
		else {
			if (i < 9) {
				cout << "    " << i + 1;
			}
			else {
				cout << "   " << i + 1;
			}
		}
	}
	cout << "\n";
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         |";
		}
		else {
			cout << "    |";
		}
	}
	cout << "\n";
}

//Imprimimos el tablero
void imprimirTablero(int *tablero, int filas, int columnas) {
	cout << "SE HAN GENERADO " << filas << " FILAS Y " << columnas << " COLUMNAS\n";
	cout << "+-+-+-TABLERO DE JUEGO-+-+-+\n\n";
	imprimirColumnas(columnas);
	for (int i = 0; i < filas; i++) {
		if (i < 9) {
			cout << i + 1 << "    - ";
		}
		else {
			cout << i + 1 << "   - ";
		}
		for (int k = 0; k < columnas; k++) {
			//Damos color en funci�n del n�mero imprimido
			int bloque = tablero[i * filas + k];
			switch (bloque) {
			case 2:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14); //Amarillo
				break;
			case 4:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 12); //Rojo
				break;
			case 8:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13); //Morado
				break;
			case 16:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 9); //Azul
				break;
			default:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7); //Blanco
			}
			if (bloque < 10) cout << "| " << bloque << " |";
			else cout << "| " << bloque << "|";
		}
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
		cout << "\n";
	}
}

//En funci�n del movimiento, llama a la comprobaci�n correspondiente
__device__ void compruebaSemillas(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	switch (movimiento){
	case 'W':
		compruebaAbajo(tablero, fila, columna, filas, columnas, movimiento);
		break;
	case 'S':
		compruebaArriba(tablero, fila, columna, filas, columnas, movimiento);
		break;
	case 'D':
		compruebaIzquierda(tablero, fila, columna, filas, columnas, movimiento);
		break;
	case 'A':
		compruebaDerecha(tablero, fila, columna, filas, columnas, movimiento);
		break;
	}

}

//Desplaza los n�meros respecto a los ceros que haya, en funci�n del movimiento
__device__ void moverCeros(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){
	if (movimiento == 'W'){
		for (int i = filas - 1; i > 0; i--){
			for (int j = i; j > 0; j--){
				if (tablero[(j * columnas) + columna] != 0 && tablero[((j - 1) * columnas) + columna] == 0){
					tablero[((j - 1) * columnas) + columna] = tablero[(j * columnas) + columna];
					tablero[(j * columnas) + columna] = 0;
				}
			}
		}
	}
	else if (movimiento == 'S'){
		for (int i = 0; i < filas - 1; i++){
			for (int j = i; j < filas - 1; j++){
				if (tablero[(j * columnas) + columna] != 0 && tablero[((j + 1) * columnas) + columna] == 0){
					tablero[((j + 1) * columnas) + columna] = tablero[(j * columnas) + columna];
					tablero[(j * columnas) + columna] = 0;
				}
			}
		}
	}
	else if (movimiento == 'D'){
		for (int i = 0; i < columnas - 1; i++){
			for (int j = i; j < columnas - 1; j++){
				if (tablero[fila * columnas + j] != 0 && tablero[fila * columnas + (j + 1)] == 0 && tablero[fila * columnas + (j + 1)] != columnas){
					tablero[fila * columnas + (j + 1)] = tablero[fila * columnas + j];
					tablero[fila * columnas + j] = 0;
				}
			}
		}
	}

	else if (movimiento == 'A'){
		for (int i = columnas - 1; i > 0; i--){
			for (int j = i; j > 0; j--){
				if (tablero[fila * columnas + j] != 0 && tablero[fila * columnas + (j - 1)] == 0){
					tablero[fila * columnas + (j - 1)] = tablero[fila * columnas + j];
					tablero[fila * columnas + j] = 0;
				}
			}
		}
	}
}

//Comprueba hacia arriba
__device__ void compruebaArriba(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[((fila - 1) * columnas) + columna]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[((fila - 1) * columnas) + columna] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}
}

//Comprueba hacia abajo
__device__ void compruebaAbajo(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[((fila + 1) * columnas) + columna]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[((fila + 1) * columnas) + columna] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}

}

//Comprueba hacia la derecha
__device__ void compruebaDerecha(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[(fila * columnas) + (columna + 1)]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[(fila * columnas) + (columna + 1)] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}

}

//Comprueba hacia la izquierda
__device__ void compruebaIzquierda(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[(fila * columnas) + (columna - 1)]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[(fila * columnas) + (columna - 1)] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}

}

__global__ void juegoManual(int *tablero, int filas, int columnas, char movimiento){

	//Guardamos la columna y la fila del hilo
	int columnaHilo = threadIdx.x;
	int filaHilo = threadIdx.y;

	compruebaSemillas(tablero, filaHilo, columnaHilo, filas, columnas, movimiento);

	__syncthreads();

}

//Guarda la partida con el tablero, las filas, las columnas y la dificultad
void guardarPartida(int *tablero, int filas, int columnas, int dificultad) {
	ofstream doc;
	doc.open("partida.txt");
	doc << filas << "\n";
	doc << columnas << "\n";
	doc << dificultad << "\n";
	for (int i = 0; i < filas * columnas; i++) {
		doc << tablero[i] << " ";
	}
	doc.close();
	system("cls");
	cout << "Guardado correctamente.\n\n";
}

//Carga la partida guardada
void cargarPartida() {

	const string fichero = "partida.txt";
	ifstream leer;
	leer.open(fichero.c_str());
	int  d, *tablero;
	int i = 0;
	int n = 48;
	int f = 0;
	int c = 0;
	char fila[80];
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		while (n > 47 && n < 58) {
			n = (int)fila[i];
			i++;
			if (n > 47 && n < 58) {
				f = f * 10 + (n - 48);
			}
		}

	}
	n = 48;
	i = 0;
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		while (n > 47 && n < 58) {
			n = (int)fila[i];
			i++;
			if (n > 47 && n < 58) {
				c = c * 10 + (n - 48);
			}
		}

	}
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		d = (int)fila[0] - 48;
	}


	tablero = new int[f*c];
	for (int i = 0; i < f * c; i++) {
		leer.getline(fila, 80, ' ');
		tablero[i] = (int)fila[0] - 48;
	}
	leer.close();
	modoManual(tablero, f, c, d);
}

void modoManual(int *tablero, int filas, int columnas, int dificultad){

	//system("cls");
	char movimiento = ' ';
	bool salida = false;
	while (movimiento != 'Z' && salida == false){
		imprimirTablero(tablero, filas, columnas);
		cout << "Pulsa W, A, S o D para mover los numeros (Z para salir): \n";
		cin >> movimiento;
		//while (movimiento != (ARRIBA || ABAJO || IZQUIERDA || DERECHA)) {
		while (movimiento != 'W' && movimiento != 'S' && movimiento != 'A' && movimiento != 'D' && movimiento != 'Z') {
			cout << "Tecla no valida, introduzca una valida:\n";
			cin >> movimiento;
		}

		//CUDA
		int *tablero_gpu;
		//Reservamos memoria y copiamos tablero en GPU
		hipMalloc((void**)&tablero_gpu, (filas * columnas) * sizeof(int));
		hipMemcpy(tablero_gpu, tablero, (filas * columnas) * sizeof(int), hipMemcpyHostToDevice);
		//Creamos los hilos en un solo bloque
		dim3 DimGrid(1, 1);
		dim3 DimBlock(filas, columnas);
		juegoManual << < DimGrid, DimBlock >> > (tablero_gpu, filas, columnas, movimiento);
		hipMemcpy(tablero, tablero_gpu, sizeof(int)* filas * columnas, hipMemcpyDeviceToHost);
		//system("cls");
		comprobarLleno(tablero, filas, columnas, dificultad, salida);
		hipFree(tablero_gpu);

	}
	//system("cls");
	cout << "Deseas guardar la partida? (S/N)\n";
	char guardar = 'x';
	cin >> guardar;
	while (guardar != 'S' && guardar != 'N') {
		system("cls");
		cout << "Valor no valido, quieres guardar la partida? (S/N): \n";
		cin >> guardar;
	}
	if (guardar == 'S') {
		guardarPartida(tablero, filas, columnas, dificultad);
	}
	else {
		cout << "Saliendo sin guardar...\n \n";
	}
}