#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <conio.h>
#include <time.h>
#include <stdio.h>
#include <fstream>
#include <windows.h>

#define ARRIBA 72
#define ABAJO 80
#define DERECHA 77
#define IZQUIERDA 75

using namespace std;

//Funciones que van a utilizarse a lo largo del programa
//CPU
void generarTablero(int *tablero, int filas, int columnas);
void imprimirTablero(int *tablero, int filas, int columnas);
void imprimirColumnas(int columnas);
void generarSemillas(int *tablero, int filas, int columnas);

//GPU
__device__ void compruebaArriba(int *tablero, int fila, int columna, int filas, int columnas, int anterior);
__device__ void compruebaAbajo(int *tablero, int fila, int columna, int filas, int columnas, int anterior);
__device__ void compruebaDerecha(int *tablero, int fila, int columna, int filas, int columnas, int anterior);
__device__ void compruebaIzquierda(int *tablero, int fila, int columna, int filas, int columnas, int anterior);

int main(void){

	//Almacenamos las propiedades de la tarjeta para no exceder el numero de hilos posibles en el tablero
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	//Propiedades del tablero
	int *tablero;
	int filas = 0;
	int columnas = 0;
	int dificultad = 0;

	//Recogemos los datos de filas y columnas del tablero que vamos a usar
	cout << "Seleccione el numero de filas con las que desea jugar: \n";
	cin >> filas;
	cout << "Seleccione el numero de columnas con las que desea jugar: \n";
	cin >> columnas;

	//Tablero m�nimo de 4 por 4
	while (filas < 4) {
		cout << "El numero de filas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
		cin >> filas;
	}
	while (columnas < 4) {
		cout << "El numero de columnas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
		cin >> columnas;
	}
	while (prop.maxThreadsPerBlock < (filas * columnas)) {
		cout << "Has excedido el limite de semillas posibles para el tablero, introduce las filas y las columnas de nuevo: \n";
		cout << "Seleccione el numero de filas con las que desea jugar: \n";
		cin >> filas;
		cout << "Seleccione el numero de columnas con las que desea jugar: \n";
		cin >> columnas;
	}

	//Reservamos la memoria del tablero y lo inicializamos con generar tablero
	tablero = new int[filas * columnas];
	generarTablero(tablero, filas, columnas);
	imprimirTablero(tablero, filas, columnas);

    
	system("PAUSE");
}

//Generar tablero con n�meros aleatorios
void generarTablero(int *tablero, int filas, int columnas){
	srand(time(0));
	int tama�o = filas * columnas;
	for (int i = 0; i < tama�o; i++){
		tablero[i] = 0;
	}
	generarSemillas(tablero, filas, columnas);
}

//Genera los n�meros para jugar en el tablero
void generarSemillas(int *tablero, int filas, int columnas){
	int tama�o = filas * columnas;
	int contador = 0;
	while (contador < 3){
		int aux = rand() % 3;
		int i = rand() % tama�o;
		//cout << "POSICION: " << i+1 << "\n";
		if (tablero[i] == 0){
			switch (aux){
			case 0:
				tablero[i] = 2;
				break;
			case 1:
				tablero[i] = 4;
				break;
			case 2:
				tablero[i] = 8;
				break;
			}
			contador++;
		}
	}
}

//Funci�n que imprime el n�mero de columnas que va a tener el tablero para que sea m�s facil elegir semillas
void imprimirColumnas(int columnas) {
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         " << i + 1;
		}
		else {
			if (i < 9) {
				cout << "    " << i + 1;
			}
			else {
				cout << "   " << i + 1;
			}
		}
	}
	cout << "\n";
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         |";
		}
		else {
			cout << "    |";
		}
	}
	cout << "\n";
}

//Imprimimos el tablero
void imprimirTablero(int *tablero, int filas, int columnas) {
	cout << "SE HAN GENERADO " << filas << " FILAS Y " << columnas << " COLUMNAS\n";
	cout << "+-+-+-TABLERO DE JUEGO-+-+-+\n\n";
	imprimirColumnas(columnas);
	for (int i = 0; i < filas; i++) {
		if (i < 9) {
			cout << i + 1 << "    - ";
		}
		else {
			cout << i + 1 << "   - ";
		}
		for (int k = 0; k < columnas; k++) {
			//Damos color en funci�n del n�mero imprimido
			int bloque = tablero[i * filas + k];
			switch (bloque) {
				case 2:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14); //Amarillo
					break;
				case 4:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 12); //Rojo
					break;
				case 8:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13); //Morado
					break;
				default:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7); //Gris
			}
			cout << "| " << bloque << " |";
		}
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
		cout << "\n";
	}
}

__device__ void compruebaSemillas(int *tablero, int fila, int columna, int filas, int columnas, int anterior){



}

__device__ void compruebaArriba(int *tablero, int fila, int columna, int filas, int columnas, int anterior){

	if (tablero[((fila - 1) * columnas) + columna] == anterior){
		tablero[((fila - 1) * columnas) + columna] = anterior * 2;
		tablero[(fila * columnas) + columna] = 0;
	}
	else if (tablero[((fila - 1) * columnas) + columna] == 0){
		tablero[((fila - 1) * columnas) + columna] = anterior;
		tablero[(fila * columnas) + columna] = 0;
	}

}

__device__ void compruebaAbajo(int *tablero, int fila, int columna, int filas, int columnas, int anterior){

	if (tablero[((fila + 1) * columnas) + columna] == anterior){
		tablero[((fila + 1) * columnas) + columna] = anterior * 2;
		tablero[(fila * columnas) + columna] = 0;
	}
	else if (tablero[((fila + 1) * columnas) + columna] == 0){
		tablero[((fila + 1) * columnas) + columna] = anterior;
		tablero[(fila * columnas) + columna] = 0;
	}

}

__device__ void compruebaDerecha(int *tablero, int fila, int columna, int filas, int columnas, int anterior){

	if (tablero[(fila * columnas) + (columna + 1)] == anterior){
		tablero[(fila * columnas) + (columna + 1)] = anterior * 2;
		tablero[(fila * columnas) + columna] = 0;
	}
	else if (tablero[(fila * columnas) + (columna + 1)] == 0){
		tablero[(fila * columnas) + (columna + 1)] = anterior;
		tablero[(fila * columnas) + columna] = 0;
	}

}

__device__ void compruebaIzquierda(int *tablero, int fila, int columna, int filas, int columnas, int anterior){

	if (tablero[(fila * columnas) + (columna - 1)] == anterior){
		tablero[(fila * columnas) + (columna - 1)] = anterior * 2;
		tablero[(fila * columnas) + columna] = 0;
	}
	else if (tablero[(fila * columnas) + (columna - 1)] == 0){
		tablero[(fila * columnas) + (columna - 1)] = anterior;
		tablero[(fila * columnas) + columna] = 0;
	}

}