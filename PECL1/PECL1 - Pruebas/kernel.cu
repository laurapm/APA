#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <conio.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <windows.h>

using namespace std;

//Funciones que van a utilizarse a lo largo del programa
//CPU
void generarTablero(int *tablero, int filas, int columnas);
void imprimirTablero(int *tablero, int filas, int columnas);
void imprimirColumnas(int columnas);
void generarSemillas(int *tablero, int filas, int columnas);
void guardarPartida(int *tablero, int filas, int columnas/*, int dificultad*/);
void cargarPartida();
void modoManual(int *tablero, int filas, int columnas);

//GPU
__global__ void juegoManual(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaSemillas(int *tablero, int filas, int columnas, char movimiento);
__device__ void compruebaArriba(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaAbajo(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaDerecha(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
__device__ void compruebaIzquierda(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);
//AUX
__device__ void moverCeros(int *tablero, int fila, int columna, int filas, int columnas, char movimiento);

int main(void){

	//Almacenamos las propiedades de la tarjeta para no exceder el numero de hilos posibles en el tablero
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	//Propiedades del tablero
	int *tablero;
	int filas = 0;
	int columnas = 0;
	int dificultad = 0;

	//Preguntamos si quiere cargar un juego guardado anteriormente o si quiere empezar de nuevo
	cout << "Quieres continuar una partida anterior o empezar de nuevo? (C: Cargar / N: Nueva partida)\n";
	char partida = 'X';
	cin >> partida;
	while (partida != 'C' && partida != 'N') {
		cout << "Introduce un valor valido para iniciar el juego\n";
		cin >> partida;
	}
	if (partida == 'N'){
		//Recogemos los datos de filas y columnas del tablero que vamos a usar
		cout << "Seleccione el numero de filas con las que desea jugar: \n";
		cin >> filas;
		cout << "Seleccione el numero de columnas con las que desea jugar: \n";
		cin >> columnas;

		//Tablero m�nimo de 4 por 4
		while (filas < 4) {
			cout << "El numero de filas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
			cin >> filas;
		}
		while (columnas < 4) {
			cout << "El numero de columnas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
			cin >> columnas;
		}
		while (prop.maxThreadsPerBlock < (filas * columnas)) {
			cout << "Has excedido el limite de semillas posibles para el tablero, introduce las filas y las columnas de nuevo: \n";
			cout << "Seleccione el numero de filas con las que desea jugar: \n";
			cin >> filas;
			cout << "Seleccione el numero de columnas con las que desea jugar: \n";
			cin >> columnas;
		}

		//Reservamos la memoria del tablero y lo inicializamos con generar tablero
		tablero = new int[filas * columnas];
		generarTablero(tablero, filas, columnas);
		modoManual(tablero, filas, columnas);

	}
	else {
		cargarPartida();
	}
	system("PAUSE");
}

//Generar tablero con n�meros aleatorios
void generarTablero(int *tablero, int filas, int columnas){
	srand(time(0));
	int tama�o = filas * columnas;
	for (int i = 0; i < tama�o; i++){
		tablero[i] = 0;
	}
	generarSemillas(tablero, filas, columnas);
}

//Genera los n�meros para jugar en el tablero
void generarSemillas(int *tablero, int filas, int columnas){
	int tama�o = filas * columnas;
	int contador = 0;
	while (contador < 3){
		int aux = rand() % 3;
		int i = rand() % tama�o;
		//cout << "POSICION: " << i+1 << "\n";
		if (tablero[i] == 0){
			switch (aux){
			case 0:
				tablero[i] = 2;
				break;
			case 1:
				tablero[i] = 4;
				break;
			case 2:
				tablero[i] = 8;
				break;
			}
			contador++;
		}
	}
}

//Funci�n que imprime el n�mero de columnas que va a tener el tablero para que sea m�s facil elegir semillas
void imprimirColumnas(int columnas) {
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         " << i + 1;
		}
		else {
			if (i < 9) {
				cout << "    " << i + 1;
			}
			else {
				cout << "   " << i + 1;
			}
		}
	}
	cout << "\n";
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         |";
		}
		else {
			cout << "    |";
		}
	}
	cout << "\n";
}

//Imprimimos el tablero
void imprimirTablero(int *tablero, int filas, int columnas) {
	cout << "SE HAN GENERADO " << filas << " FILAS Y " << columnas << " COLUMNAS\n";
	cout << "+-+-+-TABLERO DE JUEGO-+-+-+\n\n";
	imprimirColumnas(columnas);
	for (int i = 0; i < filas; i++) {
		if (i < 9) {
			cout << i + 1 << "    - ";
		}
		else {
			cout << i + 1 << "   - ";
		}
		for (int k = 0; k < columnas; k++) {
			//Damos color en funci�n del n�mero imprimido
			int bloque = tablero[i * filas + k];
			switch (bloque) {
				case 2:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14); //Amarillo
					break;
				case 4:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 12); //Rojo
					break;
				case 8:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13); //Morado
					break;
				case 16:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 9); //Azul
					break;
				default:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7); //Blanco
			}
			if (bloque < 10) cout << "| " << bloque << " |";
			else cout << "| " << bloque << "|";
		}
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
		cout << "\n";
	}
}

__device__ void compruebaSemillas(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	switch (movimiento){
	case 'W':
		compruebaAbajo(tablero, fila, columna, filas, columnas, movimiento);
		break;
	case 'S':
		compruebaArriba(tablero, fila, columna, filas, columnas, movimiento);
		break;
	case 'D':
		compruebaIzquierda(tablero, fila, columna, filas, columnas, movimiento);
		break;
	case 'A':
		compruebaDerecha(tablero, fila, columna, filas, columnas, movimiento);
		break;
	}

}


__device__ void moverCeros(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	for (int i = filas - 1; i > 0; i--){
		for (int j = i; j > 0; j--){
			if (tablero[(j * columnas) + columna] != 0 && tablero[((j - 1) * columnas) + columna] == 0){
				tablero[((j - 1) * columnas) + columna] = tablero[(j * columnas) + columna];
				tablero[(j * columnas) + columna] = 0;
			}
		}
	}

	/*
	for (int i = filas - 1; i > 0; i--){
		if (tablero[(i * columnas) + columna] == 0){
			tablero[(i * columnas) + columna] = tablero[((i - 1) * columnas) + columna];
			tablero[((i - 1) * columnas) + columna] = 0;
		}

	}*/
	/*for (int i = filas - 1; i > 0; i--){
	if (tablero[((i - 1) * columnas) + columna] != 0){
	int a = i;
	while (tablero[((a - 1) * columnas) + columna] == 0){
	tablero[(a * columnas) + columna] = tablero[((a - 1) * columnas) + columna];
	tablero[((a - 1) * columnas) + columna] = 0;
	}
	}
	}*/

}

__device__ void compruebaArriba(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[((fila - 1) * columnas) + columna]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[((fila - 1) * columnas) + columna] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}
	//compruebaArriba(tablero, fila - 1, columna, filas, columnas);
}

__device__ void compruebaAbajo(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[((fila + 1) * columnas) + columna]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[((fila + 1) * columnas) + columna] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}

}

__device__ void compruebaDerecha(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[(fila * columnas) + (columna + 1)]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[(fila * columnas) + (columna + 1)] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}

}

__device__ void compruebaIzquierda(int *tablero, int fila, int columna, int filas, int columnas, char movimiento){

	moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	if (tablero[(fila * columnas) + columna] != 0 && tablero[(fila * columnas) + columna] == tablero[(fila * columnas) + (columna - 1)]){
		tablero[(fila * columnas) + columna] = tablero[(fila * columnas) + columna] * 2;
		tablero[(fila * columnas) + (columna - 1)] = 0;
		moverCeros(tablero, fila, columna, filas, columnas, movimiento);
	}

}

__global__ void juegoManual(int *tablero, int filas, int columnas, char movimiento){

	//Guardamos la columna y la fila del hilo
	int columnaHilo = threadIdx.x;
	int filaHilo = threadIdx.y;

	compruebaSemillas(tablero, filaHilo, columnaHilo, filas, columnas, movimiento);

	__syncthreads();

}

void guardarPartida(int *tablero, int filas, int columnas/*, int dificultad*/) {
	ofstream doc;
	doc.open("partida.txt");
	doc << filas << "\n";
	doc << columnas << "\n";
	//doc << dificultad << "\n";
	for (int i = 0; i < filas * columnas; i++) {
		doc << tablero[i] << " ";
	}
	doc.close();
	system("cls");
	cout << "Guardado correctamente.\n\n";
}

void cargarPartida() { //NO FUNCIONA LE�E

	const string fichero = "partida.txt";
	ifstream leer;
	leer.open(fichero.c_str());
	int  d, *tablero;
	int i = 0;
	int n = 48;
	int f = 0;
	int c = 0;
	char fila[80];
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		while (n > 47 && n < 58) {
			n = (int)fila[i];
			i++;
			if (n > 47 && n < 58) {
				f = f * 10 + (n - 48);
			}
		}

	}
	n = 48;
	i = 0;
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		while (n > 47 && n < 58) {
			n = (int)fila[i];
			i++;
			if (n > 47 && n < 58) {
				c = c * 10 + (n - 48);
			}
		}

	}
	if (!leer.fail()) {
		leer.getline(fila, 80, '\n');
		d = (int)fila[0] - 48;
	}


	tablero = new int[f*c];
	for (int i = 0; i < f * c; i++) {
		leer.getline(fila, 80, ' ');
		tablero[i] = (int)fila[0] - 48;
	}
	leer.close();
	modoManual(tablero, f, c);
}

void modoManual(int *tablero, int filas, int columnas){

	//system("cls");
	char movimiento = ' ';
	while (movimiento != 'Z'){
		imprimirTablero(tablero, filas, columnas);
		cout << "Pulsa W, A, S o D para mover los numeros (Z para salir): \n";
		cin >> movimiento;
		//while (movimiento != (ARRIBA || ABAJO || IZQUIERDA || DERECHA)) {
		while (movimiento != 'W' && movimiento != 'S' && movimiento != 'A' && movimiento != 'D' && movimiento != 'Z') {
			cout << "Tecla no valida, introduzca una valida:\n";
			cin >> movimiento;
		}

		//CUDA
		int *tablero_gpu;
		//Reservamos memoria y copiamos tablero en GPU
		hipMalloc((void**)&tablero_gpu, (filas * columnas) * sizeof(int));
		hipMemcpy(tablero_gpu, tablero, (filas * columnas) * sizeof(int), hipMemcpyHostToDevice);
		//Creamos los hilos en un solo bloque
		dim3 DimGrid(1, 1);
		dim3 DimBlock(filas, columnas);
		juegoManual << < DimGrid, DimBlock >> > (tablero_gpu, filas, columnas, movimiento);
		hipMemcpy(tablero, tablero_gpu, sizeof(int)* filas * columnas, hipMemcpyDeviceToHost);
		//system("cls");
		generarSemillas(tablero, filas, columnas);
		hipFree(tablero_gpu);

	}
	//system("cls");
	cout << "Deseas guardar la partida? (S/N)\n";
	char guardar = 'x';
	cin >> guardar;
	while (guardar != 'S' && guardar != 'N') {
		system("cls");
		cout << "Valor no valido, quieres guardar la partida? (S/N): \n";
		cin >> guardar;
	}
	if (guardar == 'S') {
		guardarPartida(tablero, filas, columnas/*, dificultad*/);
	}
	else {
		cout << "Saliendo sin guardar...\n \n";
	}
}