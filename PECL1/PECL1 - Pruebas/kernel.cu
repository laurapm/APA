#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <conio.h>
#include <time.h>
#include <stdio.h>
#include <fstream>
#include <windows.h>

using namespace std;

void generarTablero(int *tablero, int filas, int columnas);
void imprimirTablero(int *tablero, int filas, int columnas);
void imprimirColumnas(int columnas);
void generarSemillas(int *tablero, int filas, int columnas);

int main(void){

	//Almacenamos las propiedades de la tarjeta para no exceder el numero de hilos posibles en el tablero
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	//Propiedades del tablero
	int *tablero;
	int filas = 0;
	int columnas = 0;
	int dificultad = 0;

	//Recogemos los datos de filas y columnas del tablero que vamos a usar
	cout << "Seleccione el numero de filas con las que desea jugar: \n";
	cin >> filas;
	cout << "Seleccione el numero de columnas con las que desea jugar: \n";
	cin >> columnas;

	//Tablero m�nimo de 4 por 4
	while (filas < 4) {
		cout << "El numero de filas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
		cin >> filas;
	}
	while (columnas < 4) {
		cout << "El numero de columnas con las que desea jugar es demasiado peque�o, el minimo aceptado es 4: \n";
		cin >> columnas;
	}
	while (prop.maxThreadsPerBlock < (filas * columnas)) {
		cout << "Has excedido el limite de semillas posibles para el tablero, introduce las filas y las columnas de nuevo: \n";
		cout << "Seleccione el numero de filas con las que desea jugar: \n";
		cin >> filas;
		cout << "Seleccione el numero de columnas con las que desea jugar: \n";
		cin >> columnas;
	}

	//Reservamos la memoria del tablero y lo inicializamos con generar tablero
	tablero = new int[filas * columnas];
	generarTablero(tablero, filas, columnas);
	imprimirTablero(tablero, filas, columnas);

    
	system("PAUSE");
}

//Generar tablero con n�meros aleatorios
void generarTablero(int *tablero, int filas, int columnas){
	srand(time(0));
	int tama�o = filas * columnas;
	for (int i = 0; i < tama�o; i++){
		tablero[i] = 0;
	}
	generarSemillas(tablero, filas, columnas);
}

//Genera los n�meros para jugar en el tablero
void generarSemillas(int *tablero, int filas, int columnas){
	int tama�o = filas * columnas;
	int contador = 3;
	while (contador > 0){
		int aux = rand() % 3;
		int i = rand() % tama�o + 1;
		cout << "POSICION: " << i;
		if (tablero[i] == 0){
			switch (aux){
			case 0:
				tablero[i] = 2;
				break;
			case 1:
				tablero[i] = 4;
				break;
			case 2:
				tablero[i] = 8;
				break;
			}
			contador--;
		}
	}
}

//Funci�n que imprime el n�mero de columnas que va a tener el tablero para que sea m�s facil elegir semillas
void imprimirColumnas(int columnas) {
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         " << i + 1;
		}
		else {
			if (i < 9) {
				cout << "    " << i + 1;
			}
			else {
				cout << "   " << i + 1;
			}
		}
	}
	cout << "\n";
	for (int i = 0; i < columnas; i++) {
		if (i == 0) {
			cout << "         |";
		}
		else {
			cout << "    |";
		}
	}
	cout << "\n";
}

//Imprimimos el tablero
void imprimirTablero(int *tablero, int filas, int columnas) {
	cout << "SE HAN GENERADO " << filas << " FILAS Y " << columnas << " COLUMNAS\n";
	cout << "+-+-+-TABLERO DE JUEGO-+-+-+\n\n";
	imprimirColumnas(columnas);
	for (int i = 0; i < filas; i++) {
		if (i < 9) {
			cout << i + 1 << "    - ";
		}
		else {
			cout << i + 1 << "   - ";
		}
		for (int k = 0; k < columnas; k++) {
			//Damos color en funci�n del n�mero imprimido
			int bloque = tablero[i * filas + k];
			switch (bloque) {
				case 2:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14); //Amarillo
					break;
				case 4:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 12); //Rojo
					break;
				case 8:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13); //Morado
					break;
				default:
					SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7); //Gris
			}
			cout << "| " << bloque << " |";
		}
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
		cout << "\n";
	}
}